#include "hip/hip_runtime.h"
/*
 *  _reg_mutualinformation_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_MUTUALINFORMATION_GPU_CU
#define _REG_MUTUALINFORMATION_GPU_CU

#include "_reg_blocksize_gpu.h"
#include "_reg_mutualinformation_gpu.h"
#include "_reg_mutualinformation_kernels.cu"

/// Called when we only have one target and one source image
void reg_getVoxelBasedNMIGradientUsingPW_gpu(   nifti_image *targetImage,
                                                nifti_image *resultImage,
                                                float **targetImageArray_d,
                                                float **resultImageArray_d,
                                                float4 **resultGradientArray_d,
                                                float **logJointHistogram_d,
                                                float4 **voxelNMIGradientArray_d,
                                                int **mask_d,
                                                int activeVoxelNumber,
                                                double *entropies,
                                                int refBinning,
                                                int floBinning)
{
    const int voxelNumber = targetImage->nx*targetImage->ny*targetImage->nz;
    const int binNumber = refBinning*floBinning+refBinning+floBinning;
    const float4 entropies_h=make_float4((float)entropies[0],(float)entropies[1],(float)entropies[2],(float)entropies[3]);
    const float NMI = (float)((entropies[0]+entropies[1])/entropies[2]);

    // Bind Symbols
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstTargetBin),&refBinning,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstResultBin),&floBinning,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Entropies),&entropies_h,sizeof(float4)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NMI),&NMI,sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)));

    // Texture bindingcurrentFloating
    CUDA_SAFE_CALL(hipBindTexture(0, firstTargetImageTexture, *targetImageArray_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageTexture, *resultImageArray_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageGradientTexture, *resultGradientArray_d, voxelNumber*sizeof(float4)));
    CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(*voxelNMIGradientArray_d, 0, voxelNumber*sizeof(float4)));

    const unsigned int Grid_reg_getVoxelBasedNMIGradientUsingPW =
        (unsigned int)ceil((float)activeVoxelNumber/(float)Block_reg_getVoxelBasedNMIGradientUsingPW);
    dim3 B1(Block_reg_getVoxelBasedNMIGradientUsingPW,1,1);
    dim3 G1(Grid_reg_getVoxelBasedNMIGradientUsingPW,1,1);

    reg_getVoxelBasedNMIGradientUsingPW_kernel <<< G1, B1 >>> (*voxelNMIGradientArray_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_getVoxelBasedNMIGradientUsingPW_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
}
void reg_getVoxelBasedNMIGradientUsingPW2x2_gpu(nifti_image *targetImage,
                                                nifti_image *resultImage,
                                                float **targetImageArray1_d,
                                                float **targetImageArray2_d,
                                                float **resultImageArray1_d,
                                                float **resultImageArray2_d,
                                                float4 **resultGradientArray1_d,
                                                float4 **resultGradientArray2_d,
                                                float **logJointHistogram_d,
                                                float4 **voxelNMIGradientArray_d,
                                                int **mask_d,
                                                int activeVoxelNumber,
                                                double *entropies,
                                                unsigned int *targetBinning,
                                                unsigned int *resultBinning)
{
    if (targetImage->nt != 2 || resultImage->nt != 2) {
        printf("[NiftyReg CUDA] reg_getVoxelBasedNMIGradientUsingPW2x2_gpu: This kernel should only be used with two target and source images\n");
        return;
    }
    const int voxelNumber = targetImage->nx*targetImage->ny*targetImage->nz;
    const float4 entropies_h=make_float4((float)entropies[0],(float)entropies[1],(float)entropies[2],(float)entropies[3]);
    const float NMI = (float)((entropies[0]+entropies[1])/entropies[2]);
    const int binNumber = targetBinning[0]*targetBinning[1]*resultBinning[0]*resultBinning[1] + (targetBinning[0]*targetBinning[1]) + (resultBinning[0]*resultBinning[1]);

    // Bind Symbols
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstTargetBin),&targetBinning[0],sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondTargetBin),&targetBinning[1],sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstResultBin),&resultBinning[0],sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondResultBin),&resultBinning[1],sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Entropies),&entropies_h,sizeof(float4)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NMI),&NMI,sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)));

    // Texture binding
    CUDA_SAFE_CALL(hipBindTexture(0, firstTargetImageTexture, *targetImageArray1_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, secondTargetImageTexture, *targetImageArray2_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageTexture, *resultImageArray1_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, secondResultImageTexture, *resultImageArray2_d, voxelNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageGradientTexture, *resultGradientArray1_d, voxelNumber*sizeof(float4)));
    CUDA_SAFE_CALL(hipBindTexture(0, secondResultImageGradientTexture, *resultGradientArray2_d, voxelNumber*sizeof(float4)));
    CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(*voxelNMIGradientArray_d, 0, voxelNumber*sizeof(float4)));

    const unsigned int Grid_reg_getVoxelBasedNMIGradientUsingPW2x2 =
        (unsigned int)ceil((float)activeVoxelNumber/(float)Block_reg_getVoxelBasedNMIGradientUsingPW2x2);
    dim3 B1(Block_reg_getVoxelBasedNMIGradientUsingPW2x2,1,1);
    dim3 G1(Grid_reg_getVoxelBasedNMIGradientUsingPW2x2,1,1);

    reg_getVoxelBasedNMIGradientUsingPW_kernel2x2 <<< G1, B1 >>> (*voxelNMIGradientArray_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_getVoxelBasedNMIGradientUsingPW2x2_gpu: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
}

#endif
