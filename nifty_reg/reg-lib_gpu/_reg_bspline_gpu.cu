#include "hip/hip_runtime.h"
/*
 *  _reg_bspline_gpu.cu
 *  
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_BSPLINE_GPU_CU
#define _REG_BSPLINE_GPU_CU

#include "_reg_bspline_gpu.h"
#include "_reg_bspline_kernels.cu"

/* *************************************************************** */
/* *************************************************************** */

void reg_bspline_gpu(   nifti_image *controlPointImage,
                        nifti_image *targetImage,
                        float4 **controlPointImageArray_d,
                        float4 **positionFieldImageArray_d,
                        int **mask_d,
                        int activeVoxelNumber)
{
    const int voxelNumber = targetImage->nx * targetImage->ny * targetImage->nz;
    const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    const int3 targetImageDim = make_int3(targetImage->nx, targetImage->ny, targetImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);

    const int controlPointGridMem = controlPointNumber*sizeof(float4);

    const float3 controlPointVoxelSpacing = make_float3(
        controlPointImage->dx / targetImage->dx,
        controlPointImage->dy / targetImage->dy,
        controlPointImage->dz / targetImage->dz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_TargetImageDim),&targetImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointVoxelSpacing),&controlPointVoxelSpacing,sizeof(float3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)));

    CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *controlPointImageArray_d, controlPointGridMem));
    CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)));

    const unsigned int Grid_reg_freeForm_interpolatePosition =
        (unsigned int)ceil((float)activeVoxelNumber/(float)(Block_reg_freeForm_interpolatePosition));
    dim3 BlockP1(Block_reg_freeForm_interpolatePosition,1,1);
    dim3 GridP1(Grid_reg_freeForm_interpolatePosition,1,1);

    _reg_freeForm_interpolatePosition <<< GridP1, BlockP1 >>>(*positionFieldImageArray_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[DEBUG] reg_freeForm_interpolatePosition kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
        hipGetErrorString(hipGetLastError()),GridP1.x,GridP1.y,GridP1.z,BlockP1.x,BlockP1.y,BlockP1.z);
#endif
	return;
}

/* *************************************************************** */
/* *************************************************************** */

float reg_bspline_ApproxBendingEnergy_gpu(	nifti_image *controlPointImage,
						float4 **controlPointImageArray_d)
{
	const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
	const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
	const int controlPointGridMem = controlPointNumber*sizeof(float4);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointNumber),&controlPointNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
	CUDA_SAFE_CALL(hipBindTexture(0,controlPointTexture, *controlPointImageArray_d, controlPointGridMem));

	float *penaltyTerm_d;
	CUDA_SAFE_CALL(hipMalloc((void **)&penaltyTerm_d, controlPointNumber*sizeof(float)));

	const unsigned int Grid_reg_bspline_ApproxBendingEnergy = 
		(unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_ApproxBendingEnergy));
	dim3 B1(Block_reg_bspline_ApproxBendingEnergy,1,1);
	dim3 G1(Grid_reg_bspline_ApproxBendingEnergy,1,1);

	reg_bspline_ApproxBendingEnergy_kernel <<< G1, B1 >>>(penaltyTerm_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
	printf("[DEBUG] reg_bspline_ApproxBendingEnergy kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

	float *penaltyTerm_h;
	CUDA_SAFE_CALL(hipHostMalloc((void **)&penaltyTerm_h, controlPointNumber*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpy(penaltyTerm_h, penaltyTerm_d, controlPointNumber*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(penaltyTerm_d));

	double penaltyValue=0.0;
	for(int i=0;i<controlPointNumber;i++)
		penaltyValue += penaltyTerm_h[i];
	CUDA_SAFE_CALL(hipHostFree((void *)penaltyTerm_h));

	return (float)(penaltyValue/(3.0*(double)controlPointNumber));
}

/* *************************************************************** */
/* *************************************************************** */

void reg_bspline_ComputeApproximatedJacobianMap(   nifti_image *controlPointImage,
                                                    float4 **controlPointImageArray_d,
                                                    float **jacobianMap)
{
    /* Since we are using an approximation, only 27 basis values are used
        and they can be precomputed. We will store then in constant memory */
    float xBasisValues_h[27] = {-0.0138889,0.0000000,0.0138889,-0.0555556,0.0000000,0.0555556,-0.0138889,0.0000000,0.0138889,
                                -0.0555556,0.0000000,0.0555556,-0.2222222,0.0000000,0.2222222,-0.0555556,0.0000000,0.0555556,
                                -0.0138889,0.0000000,0.0138889,-0.0555556,0.0000000,0.0555556,-0.0138889,0.0000000,0.0138889};
    float yBasisValues_h[27] = {-0.0138889,-0.0555556,-0.0138889,0.0000000,0.0000000,0.0000000,0.0138889,0.0555556,0.0138889,
                                -0.0555556,-0.2222222,-0.0555556,0.0000000,0.0000000,0.0000000,0.0555556,0.2222222,0.0555556,
                                -0.0138889,-0.0555556,-0.0138889,0.0000000,0.0000000,0.0000000,0.0138889,0.0555556,0.0138889};
    float zBasisValues_h[27] = {-0.0138889,-0.0555556,-0.0138889,-0.0555556,-0.2222222,-0.0555556,-0.0138889,-0.0555556,-0.0138889,
                                0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,
                                0.0138889,0.0555556,0.0138889,0.0555556,0.2222222,0.0555556,0.0138889,0.0555556,0.0138889};
    float *xBasisValues_d, *yBasisValues_d, *zBasisValues_d;
    CUDA_SAFE_CALL(hipMalloc((void **)&xBasisValues_d, 27*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void **)&yBasisValues_d, 27*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void **)&zBasisValues_d, 27*sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpy(xBasisValues_d, xBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(yBasisValues_d, yBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(zBasisValues_d, zBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipBindTexture(0, xBasisTexture, xBasisValues_d, 27*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, yBasisTexture, yBasisValues_d, 27*sizeof(float)));
    CUDA_SAFE_CALL(hipBindTexture(0, zBasisTexture, zBasisValues_d, 27*sizeof(float)));

    // Other constant memory and texture are binded
    const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointNumber),&controlPointNumber,sizeof(int)));
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointSpacing),&controlPointSpacing, sizeof(float3)))
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim, sizeof(int3)));
    CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *controlPointImageArray_d, controlPointNumber*sizeof(float4)));

    // The Jacobian matrices need to be reoriented
    mat33 reorient;
    reorient.m[0][0]=controlPointImage->dx; reorient.m[0][1]=0.0f; reorient.m[0][2]=0.0f;
    reorient.m[1][0]=0.0f; reorient.m[1][1]=controlPointImage->dy; reorient.m[1][2]=0.0f;
    reorient.m[2][0]=0.0f; reorient.m[2][1]=0.0f; reorient.m[2][2]=controlPointImage->dz;
    mat33 spline_ijk;
    if(controlPointImage->sform_code>0){
        spline_ijk.m[0][0]=controlPointImage->sto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->sto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->sto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->sto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->sto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->sto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->sto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->sto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->sto_ijk.m[2][2];
    }
    else{
        spline_ijk.m[0][0]=controlPointImage->qto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->qto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->qto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->qto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->qto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->qto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->qto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->qto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->qto_ijk.m[2][2];
    }
    reorient=nifti_mat33_inverse(nifti_mat33_mul(spline_ijk, reorient));
    float3 temp=make_float3(reorient.m[0][0],reorient.m[0][1],reorient.m[0][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix0),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[1][0],reorient.m[1][1],reorient.m[1][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix1),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[2][0],reorient.m[2][1],reorient.m[2][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix2),&temp,sizeof(float3)));

    // The kernel is ran
    const unsigned int Grid_reg_bspline_ApproxJacobian =
        (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_ApproxJacobian));
    dim3 B1(Block_reg_bspline_ApproxJacobian,1,1);
    dim3 G1(Grid_reg_bspline_ApproxJacobian,1,1);

    reg_bspline_ApproxJacobianDeterminant_kernel <<< G1, B1 >>>(*jacobianMap);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[DEBUG] reg_bspline_ApproxJacobianDeterminant_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

    CUDA_SAFE_CALL(hipFree(xBasisValues_d));
    CUDA_SAFE_CALL(hipFree(yBasisValues_d));
    CUDA_SAFE_CALL(hipFree(zBasisValues_d));
}

/* *************************************************************** */
/* *************************************************************** */

void reg_bspline_ComputeJacobianMap(nifti_image *targetImage,
                                    nifti_image *controlPointImage,
                                    float4 **controlPointImageArray_d,
                                    float **jacobianMap)
{
    // Some constant memory variable are computed and allocated
    const int voxelNumber = targetImage->nx * targetImage->ny * targetImage->nz;
    const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    const int3 targetImageDim = make_int3(targetImage->nx, targetImage->ny, targetImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);

    const int controlPointGridMem = controlPointNumber*sizeof(float4);

    const float3 controlPointVoxelSpacing = make_float3(
        controlPointImage->dx / targetImage->dx,
        controlPointImage->dy / targetImage->dy,
        controlPointImage->dz / targetImage->dz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_TargetImageDim),&targetImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointVoxelSpacing),&controlPointVoxelSpacing,sizeof(float3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointSpacing),&controlPointSpacing, sizeof(float3)))

    // Texture binding: control point position
    CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *controlPointImageArray_d, controlPointGridMem));

    // The Jacobian matrices need to be reoriented, the affine matrix is store in constant memory
    mat33 reorient;
    reorient.m[0][0]=controlPointImage->dx; reorient.m[0][1]=0.0f; reorient.m[0][2]=0.0f;
    reorient.m[1][0]=0.0f; reorient.m[1][1]=controlPointImage->dy; reorient.m[1][2]=0.0f;
    reorient.m[2][0]=0.0f; reorient.m[2][1]=0.0f; reorient.m[2][2]=controlPointImage->dz;
    mat33 spline_ijk;
    if(controlPointImage->sform_code>0){
        spline_ijk.m[0][0]=controlPointImage->sto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->sto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->sto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->sto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->sto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->sto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->sto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->sto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->sto_ijk.m[2][2];
    }
    else{
        spline_ijk.m[0][0]=controlPointImage->qto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->qto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->qto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->qto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->qto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->qto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->qto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->qto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->qto_ijk.m[2][2];
    }
    reorient=nifti_mat33_inverse(nifti_mat33_mul(spline_ijk, reorient));
    float3 temp=make_float3(reorient.m[0][0],reorient.m[0][1],reorient.m[0][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix0),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[1][0],reorient.m[1][1],reorient.m[1][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix1),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[2][0],reorient.m[2][1],reorient.m[2][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix2),&temp,sizeof(float3)));

    // The kernel is ran
    const unsigned int Grid_reg_bspline_Jacobian =
        (unsigned int)ceil((float)voxelNumber/(float)(Block_reg_bspline_Jacobian));
    dim3 B1(Block_reg_bspline_Jacobian,1,1);
    dim3 G1(Grid_reg_bspline_Jacobian,1,1);

    reg_bspline_JacobianDeterminant_kernel <<< G1, B1 >>>(*jacobianMap);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[DEBUG] reg_bspline_JacobianDeterminant_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
}

/* *************************************************************** */
/* *************************************************************** */

float reg_bspline_ComputeJacobianPenaltyTerm_gpu(   nifti_image *targetImage,
                                                    nifti_image *controlPointImage,
                                                    float4 **controlPointImageArray_d,
                                                    bool approximate)
{
    // The Jacobian determinant will be stored into one array
    unsigned int pointNumber;
    if(approximate)
        pointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    else pointNumber = targetImage->nvox;


    float *jacobianMap_d;
    CUDA_SAFE_CALL(hipMalloc((void **)&jacobianMap_d, pointNumber*sizeof(float)));

    // The Jacobian map is computed
    if(approximate){
        reg_bspline_ComputeApproximatedJacobianMap( controlPointImage,
                                                    controlPointImageArray_d,
                                                    &jacobianMap_d);
    }
    else{
        reg_bspline_ComputeJacobianMap( targetImage,
                                        controlPointImage,
                                        controlPointImageArray_d,
                                        &jacobianMap_d);
    }

    // The Jacobian map is transfered back to the CPU and summed over
    float *jacobianMap_h;
    CUDA_SAFE_CALL(hipHostMalloc((void **)&jacobianMap_h, pointNumber*sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpy(jacobianMap_h, jacobianMap_d, pointNumber*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(jacobianMap_d));

    double penaltyValue=0.0;
    double logTerm;
    for(int i=0;i<pointNumber;i++){
        logTerm = log((double)jacobianMap_h[i]);
        penaltyValue += logTerm*logTerm;
    }
    CUDA_SAFE_CALL(hipHostFree((void *)jacobianMap_h));

    return (float)(penaltyValue/(double)pointNumber);
}

/* *************************************************************** */
/* *************************************************************** */

void reg_bspline_ApproxBendingEnergyGradient_gpu(   nifti_image *targetImage,
                                                    nifti_image *controlPointImage,
                                                    float4 **controlPointImageArray_d,
                                                    float4 **nodeNMIGradientArray_d,
                                                    float bendingEnergyWeight)
{
	const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
	const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
	const int controlPointGridMem = controlPointNumber*sizeof(float4);

	bendingEnergyWeight *= targetImage->nx*targetImage->ny*targetImage->nz
    		/ ( controlPointImage->nx*controlPointImage->ny*controlPointImage->nz );

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointNumber),&controlPointNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Weight),&bendingEnergyWeight,sizeof(float)));
	CUDA_SAFE_CALL(hipBindTexture(0,controlPointTexture, *controlPointImageArray_d, controlPointGridMem));

	float3 *bendingEnergyValue_d;
	CUDA_SAFE_CALL(hipMalloc((void **)&bendingEnergyValue_d, 6*controlPointNumber*sizeof(float3)));
	CUDA_SAFE_CALL(hipMemset(bendingEnergyValue_d, 0, 6*controlPointNumber*sizeof(float3)));

	const unsigned int Grid_reg_bspline_storeApproxBendingEnergy =
		(unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_storeApproxBendingEnergy));
	dim3 B1(Block_reg_bspline_storeApproxBendingEnergy,1,1);
	dim3 G1(Grid_reg_bspline_storeApproxBendingEnergy,1,1);

	reg_bspline_storeApproxBendingEnergy_kernel <<< G1, B1 >>>(bendingEnergyValue_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
	printf("[DEBUG] reg_bspline_storeApproxBendingEnergy kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

	float normal[3],first[3],second[3];
	normal[0] = 1.0f/6.0f;normal[1] = 2.0f/3.0f;normal[2] = 1.0f/6.0f;
	first[0] = -0.5f;first[1] = 0.0f;first[2] = 0.5f;
	second[0] = 1.0f;second[1] = -2.0f;second[2] = 1.0f;
	
	float4 *basis_a;CUDA_SAFE_CALL(hipHostMalloc((void **)&basis_a, 27*sizeof(float4)));
	float2 *basis_b;CUDA_SAFE_CALL(hipHostMalloc((void **)&basis_b, 27*sizeof(float2)));
	short coord=0;
	for(int c=0; c<3; c++){
		for(int b=0; b<3; b++){
			for(int a=0; a<3; a++){
				basis_a[coord].x=second[a]*normal[b]*normal[c];	// z * y * x"
				basis_a[coord].y=normal[a]*second[b]*normal[c];	// z * y"* x
				basis_a[coord].z=normal[a]*normal[b]*second[c];	// z"* y * x
				basis_a[coord].w=first[a]*first[b]*normal[c];	// z * y'* x'
				basis_b[coord].x=normal[a]*first[b]*first[c];	// z'* y'* x
				basis_b[coord].y=first[a]*normal[b]*first[c];	// z'* y * x'
				coord++;
			}
		}
	}
	float4 *basis_a_d;CUDA_SAFE_CALL(hipMalloc((void **)&basis_a_d,27*sizeof(float4)));
	float2 *basis_b_d;CUDA_SAFE_CALL(hipMalloc((void **)&basis_b_d,27*sizeof(float2)));
	CUDA_SAFE_CALL(hipMemcpy(basis_a_d, basis_a, 27*sizeof(float4), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(basis_b_d, basis_b, 27*sizeof(float2), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipHostFree((void *)basis_a));
	CUDA_SAFE_CALL(hipHostFree((void *)basis_b));
	CUDA_SAFE_CALL(hipBindTexture(0, basisValueATexture, basis_a_d, 27*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, basisValueBTexture, basis_b_d, 27*sizeof(float2)));

	const unsigned int Grid_reg_bspline_getApproxBendingEnergyGradient =
		(unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_getApproxBendingEnergyGradient));
	dim3 B2(Block_reg_bspline_getApproxBendingEnergyGradient,1,1);
	dim3 G2(Grid_reg_bspline_getApproxBendingEnergyGradient,1,1);

	reg_bspline_getApproxBendingEnergyGradient_kernel <<< G2, B2 >>>(	bendingEnergyValue_d,
										                                *nodeNMIGradientArray_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
	printf("[DEBUG] reg_bspline_getApproxBendingEnergyGradient kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G2.x,G2.y,G2.z,B2.x,B2.y,B2.z);
#endif

	CUDA_SAFE_CALL(hipFree((void *)basis_a_d));
	CUDA_SAFE_CALL(hipFree((void *)basis_b_d));
	CUDA_SAFE_CALL(hipFree((void *)bendingEnergyValue_d));

	return;
}

/* *************************************************************** */
/* *************************************************************** */

void reg_bspline_ComputeJacobianGradient_gpu(   nifti_image *targetImage,
                                                nifti_image *controlPointImage,
                                                float4 **controlPointImageArray_d,
                                                float4 **nodeNMIGradientArray_d,
                                                float jacobianWeight,
                                                bool approximate)
{
    // The Jacobian matrices need to be reoriented
    mat33 reorient;
    reorient.m[0][0]=controlPointImage->dx; reorient.m[0][1]=0.0f; reorient.m[0][2]=0.0f;
    reorient.m[1][0]=0.0f; reorient.m[1][1]=controlPointImage->dy; reorient.m[1][2]=0.0f;
    reorient.m[2][0]=0.0f; reorient.m[2][1]=0.0f; reorient.m[2][2]=controlPointImage->dz;
    mat33 spline_ijk;
    if(controlPointImage->sform_code>0){
        spline_ijk.m[0][0]=controlPointImage->sto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->sto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->sto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->sto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->sto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->sto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->sto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->sto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->sto_ijk.m[2][2];
    }
    else{
        spline_ijk.m[0][0]=controlPointImage->qto_ijk.m[0][0];
        spline_ijk.m[0][1]=controlPointImage->qto_ijk.m[0][1];
        spline_ijk.m[0][2]=controlPointImage->qto_ijk.m[0][2];
        spline_ijk.m[1][0]=controlPointImage->qto_ijk.m[1][0];
        spline_ijk.m[1][1]=controlPointImage->qto_ijk.m[1][1];
        spline_ijk.m[1][2]=controlPointImage->qto_ijk.m[1][2];
        spline_ijk.m[2][0]=controlPointImage->qto_ijk.m[2][0];
        spline_ijk.m[2][1]=controlPointImage->qto_ijk.m[2][1];
        spline_ijk.m[2][2]=controlPointImage->qto_ijk.m[2][2];
    }
    reorient=nifti_mat33_inverse(nifti_mat33_mul(spline_ijk, reorient));
    float3 temp=make_float3(reorient.m[0][0],reorient.m[0][1],reorient.m[0][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix0),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[1][0],reorient.m[1][1],reorient.m[1][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix1),&temp,sizeof(float3)));
    temp=make_float3(reorient.m[2][0],reorient.m[2][1],reorient.m[2][2]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_AffineMatrix2),&temp,sizeof(float3)));

    // Constant memory allocation
    const int voxelNumber = targetImage->nx * targetImage->ny * targetImage->nz;
    const int controlPointNumber = controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    const int3 targetImageDim = make_int3(targetImage->nx, targetImage->ny, targetImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);

    const float3 controlPointVoxelSpacing = make_float3(
        controlPointImage->dx / targetImage->dx,
        controlPointImage->dy / targetImage->dy,
        controlPointImage->dz / targetImage->dz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointNumber),&controlPointNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_TargetImageDim),&targetImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointVoxelSpacing),&controlPointVoxelSpacing,sizeof(float3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointSpacing),&controlPointSpacing, sizeof(float3)))

    // Texture binding: control point position
    CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *controlPointImageArray_d, controlPointNumber*sizeof(float4)));

    // All the values will be store in an array
    float *jacobianMatrices_d;

    if(approximate){
            /* Since we are using an approximation, only 27 basis values are used
            and they can be precomputed. We will store then in constant memory */
        float xBasisValues_h[27] = {-0.0138889,0.0000000,0.0138889,-0.0555556,0.0000000,0.0555556,-0.0138889,0.0000000,0.0138889,
                                    -0.0555556,0.0000000,0.0555556,-0.2222222,0.0000000,0.2222222,-0.0555556,0.0000000,0.0555556,
                                    -0.0138889,0.0000000,0.0138889,-0.0555556,0.0000000,0.0555556,-0.0138889,0.0000000,0.0138889};
        float yBasisValues_h[27] = {-0.0138889,-0.0555556,-0.0138889,0.0000000,0.0000000,0.0000000,0.0138889,0.0555556,0.0138889,
                                    -0.0555556,-0.2222222,-0.0555556,0.0000000,0.0000000,0.0000000,0.0555556,0.2222222,0.0555556,
                                    -0.0138889,-0.0555556,-0.0138889,0.0000000,0.0000000,0.0000000,0.0138889,0.0555556,0.0138889};
        float zBasisValues_h[27] = {-0.0138889,-0.0555556,-0.0138889,-0.0555556,-0.2222222,-0.0555556,-0.0138889,-0.0555556,-0.0138889,
                                    0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,0.0000000,
                                    0.0138889,0.0555556,0.0138889,0.0555556,0.2222222,0.0555556,0.0138889,0.0555556,0.0138889};
        float *xBasisValues_d, *yBasisValues_d, *zBasisValues_d;
        CUDA_SAFE_CALL(hipMalloc((void **)&xBasisValues_d, 27*sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc((void **)&yBasisValues_d, 27*sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc((void **)&zBasisValues_d, 27*sizeof(float)));
        CUDA_SAFE_CALL(hipMemcpy(xBasisValues_d, xBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(yBasisValues_d, yBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(zBasisValues_d, zBasisValues_h, 27*sizeof(float), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipBindTexture(0, xBasisTexture, xBasisValues_d, 27*sizeof(float)));
        CUDA_SAFE_CALL(hipBindTexture(0, yBasisTexture, yBasisValues_d, 27*sizeof(float)));
        CUDA_SAFE_CALL(hipBindTexture(0, zBasisTexture, zBasisValues_d, 27*sizeof(float)));

        CUDA_SAFE_CALL(hipMalloc((void **)&jacobianMatrices_d,
            10*(controlPointImage->nx-2)*(controlPointImage->ny-2)*(controlPointImage->nz-2)*sizeof(float)));

        // The Jacobian matrices array is filled
        const unsigned int Grid_reg_bspline_ApproxJacobian =
            (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_ApproxJacobian));
        dim3 B1(Block_reg_bspline_ApproxJacobian,1,1);
        dim3 G1(Grid_reg_bspline_ApproxJacobian,1,1);

        reg_bspline_ApproxJacobianMatrix_kernel <<< G1, B1 >>>(jacobianMatrices_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
        printf("[DEBUG] reg_bspline_ApproxJacobianMatrix_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
            hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

        CUDA_SAFE_CALL(hipFree(xBasisValues_d));
        CUDA_SAFE_CALL(hipFree(yBasisValues_d));
        CUDA_SAFE_CALL(hipFree(zBasisValues_d));
    }
    else{
        CUDA_SAFE_CALL(hipMalloc((void **)&jacobianMatrices_d,
            10*targetImage->nvox*sizeof(float)));

        // The Jacobian matrices array is filled
        const unsigned int Grid_reg_bspline_Jacobian =
            (unsigned int)ceil((float)targetImage->nvox/(float)(Block_reg_bspline_Jacobian));
        dim3 B1(Block_reg_bspline_Jacobian,1,1);
        dim3 G1(Grid_reg_bspline_Jacobian,1,1);

        reg_bspline_JacobianMatrix_kernel <<< G1, B1 >>>(jacobianMatrices_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
        printf("[DEBUG] reg_bspline_JacobianMatrix_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
            hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
    }

    // The gradient computed on a node basis
    if(approximate){
        // The weight is transfered to constant memory
        float weight=jacobianWeight;
        weight = jacobianWeight * targetImage->nvox
            / ( controlPointImage->nx*controlPointImage->ny*controlPointImage->nz);
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Weight),&weight,sizeof(float)));

        // The jacobian matrices are binded to a texture
        CUDA_SAFE_CALL(hipBindTexture(0, jacobianDeterminantTexture, jacobianMatrices_d,
            10*(controlPointImage->nx-2)*(controlPointImage->ny-2)*(controlPointImage->nz-2)*sizeof(float)));

        const unsigned int Grid_reg_bspline_ApproxJacobianGradient =
            (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_ApproxJacobianGradient));
        dim3 B2(Block_reg_bspline_ApproxJacobianGradient,1,1);
        dim3 G2(Grid_reg_bspline_ApproxJacobianGradient,1,1);

        reg_bspline_ApproxJacobianGradient_kernel <<< G2, B2 >>>(*nodeNMIGradientArray_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
        printf("[DEBUG] reg_bspline_ApproxJacobianGradient_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
            hipGetErrorString(hipGetLastError()),G2.x,G2.y,G2.z,B2.x,B2.y,B2.z);
#endif
    }
    else{
        // The weight is transfered to constant memory
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Weight),&jacobianWeight,sizeof(float)));

        // The jacobian matrices are binded to a texture
        CUDA_SAFE_CALL(hipBindTexture(0, jacobianDeterminantTexture, jacobianMatrices_d,
            10*targetImage->nvox*sizeof(float)));

        const unsigned int Grid_reg_bspline_JacobianGradient =
            (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_JacobianGradient));
        dim3 B2(Block_reg_bspline_JacobianGradient,1,1);
        dim3 G2(Grid_reg_bspline_JacobianGradient,1,1);

        reg_bspline_JacobianGradient_kernel <<< G2, B2 >>>(*nodeNMIGradientArray_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
        printf("[DEBUG] reg_bspline_JacobianGradient_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
            hipGetErrorString(hipGetLastError()),G2.x,G2.y,G2.z,B2.x,B2.y,B2.z);
#endif
    }

    CUDA_SAFE_CALL(hipFree(jacobianMatrices_d));
}

/* *************************************************************** */
/* *************************************************************** */

float reg_bspline_correctFolding_gpu(   nifti_image *targetImage,
                                        nifti_image *controlPointImage,
                                        float4 **controlPointImageArray_d,
                                        bool approx)
{
    // The Jacobian determinant will be stored into one array
    const unsigned int controlPointNumber =
        controlPointImage->nx*controlPointImage->ny*controlPointImage->nz;
    const int voxelNumber = targetImage->nvox;
    float *jacobianMap;
    CUDA_SAFE_CALL(hipMalloc((void **)&jacobianMap,
        voxelNumber*sizeof(float)));


    // The Jacobian map is computed
    reg_bspline_ComputeJacobianMap( targetImage,
                                    controlPointImage,
                                    controlPointImageArray_d,
                                    &jacobianMap);

    // Some variables are allocated into the constant memory
    const int3 targetImageDim = make_int3(targetImage->nx, targetImage->ny, targetImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);

    const float3 controlPointVoxelSpacing = make_float3(
        controlPointImage->dx / targetImage->dx,
        controlPointImage->dy / targetImage->dy,
        controlPointImage->dz / targetImage->dz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_TargetImageDim),&targetImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointVoxelSpacing),&controlPointVoxelSpacing,sizeof(float3)));

    // The jacobian map is bind to a texture
    CUDA_SAFE_CALL(hipBindTexture(0, jacobianDeterminantTexture, jacobianMap, voxelNumber*sizeof(float)));

    // For all Jacobian determinant below a threshold, the control point position will be "centered"
    const unsigned int Grid_reg_bspline_correcFolding =
        (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_bspline_correcFolding));
    dim3 B1(Block_reg_bspline_correcFolding,1,1);
    dim3 G1(Grid_reg_bspline_correcFolding,1,1);
    reg_spline_correcFolding_kernel <<< G1, B1 >>>(*controlPointImageArray_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[DEBUG] reg_spline_correcFolding_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

    CUDA_SAFE_CALL(hipFree(jacobianMap));
    return 0.0f;
}

/* *************************************************************** */
/* *************************************************************** */

void reg_spline_cppComposition_gpu( nifti_image *toUpdate,
                                    nifti_image *toCompose,
                                    float4 **toUpdateArray_d,
                                    float4 **toComposeArray_d,
                                    float ratio,
                                    bool type)
{
    if(toUpdate->nvox != toCompose->nvox){
        fprintf(stderr,"ERROR:\treg_spline_cppComposition_gpu\n");
        fprintf(stderr,"ERROR:\tBoth image are expected to have the same size ... Exit\n");
        exit(1);
    }

    const int controlPointNumber = toCompose->nx*toCompose->ny*toCompose->nz;
    const int3 controlPointImageDim = make_int3(toCompose->nx, toCompose->ny, toCompose->nz);

    const int controlPointGridMem = controlPointNumber*sizeof(float4);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Type),&type,sizeof(bool)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointNumber),&controlPointNumber,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&controlPointImageDim,sizeof(int3)));

    // The transformation matrix is binded to a texture
    float4 *transformationMatrix_h;
    float4 *voxelToRealMatrix_d;
    float4 *realToVoxelMatrix_d;
    CUDA_SAFE_CALL(hipHostMalloc((void **)&transformationMatrix_h, 3*sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **)&voxelToRealMatrix_d, 3*sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **)&realToVoxelMatrix_d, 3*sizeof(float4)));
    mat44 *voxelToReal=NULL;
    mat44 *realToVoxel=NULL;
    if(toUpdate->sform_code>0){
        voxelToReal=&(toUpdate->sto_xyz);
        realToVoxel=&(toUpdate->sto_ijk);
    }
    else{
        voxelToReal=&(toUpdate->qto_xyz);
        realToVoxel=&(toUpdate->qto_ijk);
    }
    for(int i=0; i<3; i++){
        transformationMatrix_h[i].x=voxelToReal->m[i][0];
        transformationMatrix_h[i].y=voxelToReal->m[i][1];
        transformationMatrix_h[i].z=voxelToReal->m[i][2];
        transformationMatrix_h[i].w=voxelToReal->m[i][3];
    }
    CUDA_SAFE_CALL(hipMemcpy(voxelToRealMatrix_d, transformationMatrix_h, 3*sizeof(float4), hipMemcpyHostToDevice));
    hipBindTexture(0,txVoxelToRealMatrix,voxelToRealMatrix_d,3*sizeof(float4));
    for(int i=0; i<3; i++){
        transformationMatrix_h[i].x=realToVoxel->m[i][0];
        transformationMatrix_h[i].y=realToVoxel->m[i][1];
        transformationMatrix_h[i].z=realToVoxel->m[i][2];
        transformationMatrix_h[i].w=realToVoxel->m[i][3];
    }
    CUDA_SAFE_CALL(hipMemcpy(realToVoxelMatrix_d, transformationMatrix_h, 3*sizeof(float4), hipMemcpyHostToDevice));
    hipBindTexture(0,txRealToVoxelMatrix,realToVoxelMatrix_d,3*sizeof(float4));
    CUDA_SAFE_CALL(hipHostFree((void *)transformationMatrix_h));

    // The control point grid is binded to a texture
    CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *toComposeArray_d, controlPointGridMem));

    const unsigned int Grid_reg_freeForm_interpolatePosition = 
        (unsigned int)ceil((float)controlPointNumber/(float)(Block_reg_spline_cppComposition));
    dim3 BlockP1(Block_reg_freeForm_interpolatePosition,1,1);
    dim3 GridP1(Grid_reg_freeForm_interpolatePosition,1,1);

    _reg_spline_cppComposition_kernel <<< GridP1, BlockP1 >>>(*toUpdateArray_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[DEBUG] _reg_spline_cppComposition_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
           hipGetErrorString(hipGetLastError()),GridP1.x,GridP1.y,GridP1.z,BlockP1.x,BlockP1.y,BlockP1.z);
#endif
    return;
}

/* *************************************************************** */
/* *************************************************************** */


#endif
